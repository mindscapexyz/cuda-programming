#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel definition
__global__ void VecAdd(float* A, float* B, float* C)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    C[i] = A[i] + B[i];
}

int main()
{
    int N = 10240;
    size_t size = N * sizeof(float);

    // h == host == cpu
    float *h_a, *h_b, *h_c;
    
    // d == device == gpu
    float *d_a, *d_b, *d_c;
    
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c = (float*)malloc(size);
    
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }
    
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);


    // Kernel invocation with N threads
    int threadsPerBlock = 1024;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {

        if (h_c[i] != h_a[i] + h_b[i]) {
            printf("Error: %f + %f != %f\n", h_a[i], h_b[i], h_c[i]);
            break;
        }
    }
    
    free(h_a); free(h_b); free(h_c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    
    return 0;
}